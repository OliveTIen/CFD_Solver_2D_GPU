#include "BoundarySetMap.h"

void GPU::BoundarySetMap::cuda_memcpy(BoundarySetMap* dist, const BoundarySetMap* src, hipMemcpyKind kind) {
	myint num = dist->size;
	hipMemcpy(dist->type, src->type, num * sizeof(myint), kind);
}
