#include "hip/hip_runtime.h"
#include "StructColorMap.h"
#include "../../gpu/GPUGlobalFunction.h"

void GPU::ColorMap::cuda_memcpy(ColorMap* dist, const ColorMap* src, hipMemcpyKind kind) {
	int num = src->num_control_point;
	hipMemcpy(dist->data, src->data, num * sizeof(float4), kind);
}
