#include "hip/hip_runtime.h"
#include "BasicAlgorithmGPU.h"
#include "../gpu/GPUGlobalFunction.h"

__global__ void GPU::Math::vector_weighted_divide_kernel(myint length, myfloat* v1, const myfloat* v2, myfloat weight) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < length) {
        v1[i] /= weight * v2[i];// 运算优先级 先算右侧乘法，后算*=或/=
    }
}

__global__ void GPU::Math::vector_weighted_reciprocal_kernel(myint length, myfloat* v1, const myfloat* v2, myfloat weight) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < length) {
        v1[i] = weight * v2[i] / v1[i];
    }
}

__global__  void GPU::Math::vector_weighted_add_kernel(myint length, myfloat* v1, const myfloat* v2, myfloat weight) {
    myint i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < length) {
        v1[i] += weight * v2[i];
    }
}

__global__  void GPU::Math::vector_dot_product_add_kernel(myint v_size, myfloat* v1, const myfloat* v2, const myfloat* v3) {
    myint i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < v_size) {
        v1[i] += v2[i] * v3[i];
    }
}

__global__ void assign_elements_in_array_device_kernel(myint start, myint end, myfloat* arr, myfloat value) {
    const myint num = end - start;
    const myint idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num) {
        arr[start + idx] = value;
    }
}

void GPU::Math::assign_elements_in_array_device(myint start, myint end, myfloat* arr_dev, myfloat value) {
    // 给数组指定范围[start, end)的元素赋值
    myint num = end - start;
    if (num == 0)return;
    int block_size = GPU::get_max_threads_per_block();
	int grid_size = (num + block_size - 1) / block_size;
	dim3 block(block_size, 1, 1);
	dim3 grid(grid_size, 1, 1);
	assign_elements_in_array_device_kernel <<<grid, block>>> (start, end, arr_dev, value);
}

// device函数指针
__device__ func_bin_myfloat GPU::Math::p_operator_min = operator_min;
__device__ func_bin_myfloat GPU::Math::p_operator_max = operator_max;