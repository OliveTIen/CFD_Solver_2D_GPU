#include "hip/hip_runtime.h"
#include "FluxGPU.h"
#include "../legacy/FVM_2D.h"
#include "../gpu/GPUGlobalFunction.h"
#include "../math/MathGPU.h"
#include "convection/ConvectionGPU.h"
#include "../output/LogWriter.h"
#include "restrict/RestrictGPU.h"
#include "../solvers/SolverDataGetter.h"
#include "../solvers/GPUSolver2.h"
#include "../boundary_condition/CBoundaryDoubleShockReflect.h"
#include "viscous_flux/ViscousFluxGPU.h"


// 单元通量清零
inline void clear_element_flux_device() {
    GPU::GPUSolver2* solver = SolverDataGetter::getSolverInstance();
    GPU::ElementFieldSoA& elementField_device = solver->elementField_device;

    integer num = elementField_device.num;
    for (int i = 0; i < 4; i++) {
        hipMemset(elementField_device.Flux[i], 0, num * sizeof(myfloat));
    }
    
    getLastCudaError("clear_element_flux_device failed.");
}

__device__ inline bool isUpStreamOfShock_atBoundary_static_inline(double x, double y, double _shock_x, double _shock_y, double _shock_speed, double _t_RK, const double _sqrt3) {
    // CBoundaryDoubleShockReflect的成员函数的静态版本
    double right = _shock_x + (y + _shock_speed * _t_RK) / _sqrt3;
    if (x < right) return true;
    else return false;
}

__device__ inline void get_U_reconstruct_singleValue(myfloat x, myfloat y, myfloat& U_dist, myint i_e, const myfloat* x_e, const myfloat* y_e, const myfloat* U_e, const myfloat* Ux_e, const myfloat* Uy_e, int flag_reconstruct) {
    if (flag_reconstruct == _REC_constant) {
        U_dist = U_e[i_e];
    }
    else if (flag_reconstruct == _REC_linear) {
        U_dist = U_e[i_e] + Ux_e[i_e] * (x - x_e[i_e]) + Uy_e[i_e] * (y - y_e[i_e]);
    }
    else {
        printf("invalid flag_reconstruct. @get_U_reconstruct_singleValue.\n");
        U_dist = U_e[i_e];
    }
}

__device__ void get_Uvector_reconstruct_2_device(GPU::ElementSoA& element, GPU::ElementFieldSoA& elementField, myfloat x, myfloat y, myint iElement, myfloat* U_dist, int flag_reconstruct, myfloat gamma) {
    // 根据单元分布函数，得到某点U值
    for (int i = 0; i < 4; i++) {
        get_U_reconstruct_singleValue(x, y, U_dist[i], iElement, element.xy[0], element.xy[1], elementField.U[i], elementField.Ux[i], elementField.Uy[i], flag_reconstruct);
    }

    // 若数据异常，则常量重构
    myfloat ruvp[4]{};
    GPU::Math::U2ruvp_device(U_dist, ruvp, gamma);
    if (GPU::Space::Restrict::outOfRange_device(ruvp)) {
        for (int i = 0; i < 4; i++) {
            U_dist[i] = elementField.U[i][iElement];
        }
    }
}

__device__ void get_UR_wallNonViscous_device(const myfloat* U_L, myfloat* U_R, myfloat nx, myfloat ny) {
    // 滑移壁面，法向速度取反，切向速度相等
    myfloat uxL = U_L[1] / U_L[0];
    myfloat uyL = U_L[2] / U_L[0];
    myfloat unL = uxL * nx + uyL * ny;// xy为全局坐标系，tn为切向-法向坐标系
    myfloat utL = uxL * ny - uyL * nx;
    myfloat unR = -unL;
    myfloat utR = utL;
    myfloat uxR = unR * nx + utR * ny;
    myfloat uyR = unR * ny - utR * nx;
    U_R[0] = U_L[0];
    U_R[1] = U_R[0] * uxR;
    U_R[2] = U_R[0] * uyR;
    U_R[3] = U_L[3];
}

__device__ void get_UR_wall_adiabat_device(const myfloat* U_L, myfloat* U_R, myfloat nx, myfloat ny) {
    // 无滑移绝热壁面，法向切向速度都取反
    U_R[0] = U_L[0];
    U_R[1] = -U_R[1];
    U_R[2] = -U_R[2];
    U_R[3] = U_L[3];
}

__device__ void get_UR_farField_device(const myfloat* U_L, const myfloat* ruvp_inf, myfloat* U_R, myfloat nx, myfloat ny, myfloat gamma) {
    /*
    以亚声速入口为例，先由内点外推边界点的估计值，
    再建立边界点数值解与估计值的特征相容关系
    解出边界声速，进而得到压力数值解
    */
    myfloat ga1 = gamma - 1;
    myfloat rho_f = ruvp_inf[0];
    myfloat u_f = ruvp_inf[1];
    myfloat v_f = ruvp_inf[2];
    myfloat p_f = ruvp_inf[3];

    myfloat ruvp_e[4]{};
    GPU::Math::U2ruvp_device(U_L, ruvp_e, gamma);
    myfloat rho_e = ruvp_e[0];
    myfloat u_e = ruvp_e[1];
    myfloat v_e = ruvp_e[2];
    myfloat p_e = ruvp_e[3];

    // 特征相容关系
    myfloat a2_f = gamma * p_f / rho_f;
    myfloat af = sqrt(a2_f);// 远场声速
    myfloat Ma2_f = (u_f * u_f + v_f * v_f) / a2_f;// 远场马赫数平方
    myfloat ae = sqrt(gamma * p_e / rho_e);// 内点声速
    myfloat qnf = u_f * nx + v_f * ny;// 远场法向速度
    myfloat qne = u_e * nx + v_e * ny;// 内点法向速度
    myfloat rf = qnf - 2.0 * af / ga1;// 第一波特征 R2 = u0n - 2*a0n/(gamma-1)
    myfloat re = qne + 2.0 * ae / ga1;// 第二波特征 u0n - 2*a0n/(gamma-1)
    myfloat qn = 0.5 * (re + rf);
    myfloat as = ga1 * (re - rf) / 4.0;
    myfloat dnt = 0;// 壁面运动速度

    // 判据 q<=-as，超音速入口；-as<q<0，亚音速入口；0<=q<as，亚音速出口；as<=q，超音速出口
    myfloat rho0 = 1, u0 = 0, v0 = 0, p0 = 1;
    if (qn <= -as) {
        // 超音速入口
        rho0 = rho_f;
        u0 = u_f;
        v0 = v_f;
        p0 = p_f;
    }
    else if (qn < 0) {
        // -as < qn < 0 注意C++中不能连续写小于号
        // 亚音速入口
        myfloat son = p_f / pow(rho_f, gamma);// 熵
        myfloat qtx = u_f - qnf * nx;
        myfloat qty = v_f - qnf * ny;

        rho0 = pow((as * as / son / gamma), 1.0 / ga1);// 根据S=p/rho^gamma得rho=(p/S)^(1/gamma)=(a2/gamma/S)^(1/gamma)
        u0 = qtx + qn * nx;
        v0 = qty + qn * ny;
        p0 = as * as * rho0 / gamma;// p=rho*R*t=rho/gamma*gamma*R*t=rho/gamma*a2
    }
    else if (qn < as) {
        // 0 <= qn < as
        // 亚音速出口 
        myfloat son = p_e / pow(rho_e, gamma);
        myfloat qtx = u_e - qne * nx;
        myfloat qty = v_e - qne * ny;

        // 后面跟亚音速入口相同
        rho0 = pow((as * as / son / gamma), 1.0 / ga1);
        u0 = qtx + qn * nx;
        v0 = qty + qn * ny;
        p0 = as * as * rho0 / gamma;
    }
    else {
        // as <= q
        // 超音速出口
        rho0 = rho_e;
        u0 = u_e;
        v0 = v_e;
        p0 = p_e;
    }
    myfloat ruvp0[4]{ rho0,u0,v0,p0 };
    // 计算flux
    if (GPU::Space::Restrict::outOfRange_device(ruvp0)) {
        printf("Out of range @get_UR_farField \n");
    }
    GPU::Math::ruvp2U_device(ruvp0, U_R, gamma);
}

__device__ void get_UR_inner_and_periodic_device(GPU::ElementSoA& element, GPU::ElementFieldSoA& elementField, GPU::EdgeSoA& edge, myint iEdge, myint iElementR, myfloat x_edge, myfloat y_edge, myfloat* U_R, int inviscid_flux_method_flag_reconstruct, myfloat gamma) {
    if (edge.setID[iEdge] != -1) {
        // 周期边界 应使用对称的edge的坐标计算U值
        int ID = edge.ID[iEdge];
        int ID_pair = edge.periodicPair[ID];
        if (ID_pair < 0 || ID_pair >= edge.num_edge) {
            printf("periodicPairNotFoundException, @GPU::Space::Flux::getEdgeFlux_inner_and_periodic_kernel\n");
            return;
        }

        myfloat x_edge_pair = edge.xy[0][ID_pair];
        myfloat y_edge_pair = edge.xy[1][ID_pair];
        get_Uvector_reconstruct_2_device(element, elementField, x_edge_pair, y_edge_pair, iElementR, U_R, inviscid_flux_method_flag_reconstruct, gamma);
    }
    else {
        // 内部边界
        get_Uvector_reconstruct_2_device(element, elementField, x_edge, y_edge, iElementR, U_R, inviscid_flux_method_flag_reconstruct, gamma);
    }

}

__device__ void get_flux_RiemannSolve_device(const myfloat* UL, const myfloat* UR, const myfloat nx, const myfloat ny, const myfloat length, myfloat* flux, const int flux_conservation_scheme, myfloat gamma, myfloat rcpcv) {
    // 黎曼求解器，根据UL、UR求解flux
    myfloat faceNormal[2]{ nx,ny };

    switch (flux_conservation_scheme) {// GlobalPara::inviscid_flux_method::flux_conservation_scheme
    case _SOL_LocalLaxFriedrichs:
        GPU::Space::Convection::LocalLaxFriedrichs2d(UL, UR, nx, ny, length, flux, gamma);
        break;
    case _SOL_Roe:
        GPU::Space::Convection::ConvectRoeCommon2d(UL, UR, faceNormal, length, flux, gamma, rcpcv);
        break;
    default:
        printf("invalid scheme @get_flux_RiemannSolve");
        break;
    }
}

__global__ void edge_convection_flux_device_kernel(GPU::ElementSoA element_device, GPU::ElementFieldSoA elementField_device, GPU::EdgeSoA edge_device, GPU::EdgeFieldSoA edgeField_device, GPU::BoundarySetMap boundary_device,
    int flux_conservation_scheme, int inviscid_flux_method_flag_reconstruct, myfloat* inf_ruvp, myfloat* inlet_ruvp, myfloat* outlet_ruvp, myfloat gamma, myfloat rcpcv,
    myfloat dsr_shock_x, myfloat dsr_shock_y, myfloat dsr_shock_speed, myfloat dsr_t_RK, const myfloat _sqrt3) {
    

    const int iEdge = blockIdx.x * blockDim.x + threadIdx.x;
    if (iEdge >= edge_device.num_edge || iEdge < 0) return;

    int setID = edge_device.setID[iEdge];// 内部edge不属于任何set，因此setID为-1 setID的初始化见readContinueFile
    int bType = -1;
    if (setID != -1) {
        // 第iedge个edge的边界类型
        bType = boundary_device.type[setID - 1];
    }

    myfloat nx = edge_device.normal[0][iEdge];
    myfloat ny = edge_device.normal[1][iEdge];
    myfloat x_edge = edge_device.xy[0][iEdge];
    myfloat y_edge = edge_device.xy[1][iEdge];
    myfloat length = edge_device.length[iEdge];
    myint iElementL = edge_device.elementL[iEdge];
    myint iElementR = edge_device.elementR[iEdge];
    bool isUpStream_doubleShockReflect = isUpStreamOfShock_atBoundary_static_inline(x_edge, y_edge, dsr_shock_x, dsr_shock_y, dsr_shock_speed, dsr_t_RK, _sqrt3);

    myfloat U_L[4]{};
    myfloat U_R[4]{};
    myfloat flux[4]{};
    get_Uvector_reconstruct_2_device(element_device, elementField_device, x_edge, y_edge, iElementL, U_L, inviscid_flux_method_flag_reconstruct, gamma);// get U_L

    switch (bType) {
    case _BC_symmetry:
        // 对称。对欧拉方程，相当于无粘固壁
        get_UR_wallNonViscous_device(U_L, U_R, nx, ny);
        break;
    case _BC_wall_nonViscous:
        // 无粘固壁
        get_UR_wallNonViscous_device(U_L, U_R, nx, ny);
        break;
    case _BC_wall_adiabat:
        // 无滑移绝热
        get_UR_wall_adiabat_device(U_L, U_R, nx, ny);
        break;
    case _BC_inlet:
        // 入口
        get_UR_farField_device(U_L, inlet_ruvp, U_R, nx, ny, gamma);
        break;
    case _BC_outlet:
        // 出口
        get_UR_farField_device(U_L, outlet_ruvp, U_R, nx, ny, gamma);
        break;
    case _BC_inf:
        // 远场
        get_UR_farField_device(U_L, inf_ruvp, U_R, nx, ny, gamma);
        break;
    case _BC_doubleShockReflect:
        // 双马赫反射
        if (isUpStream_doubleShockReflect) {
            get_UR_farField_device(U_L, inlet_ruvp, U_R, nx, ny, gamma);
        }
        else {
            get_UR_farField_device(U_L, outlet_ruvp, U_R, nx, ny, gamma);
        }
        break;

    default:// 内部：bType=-1，边界：bType取_BC_periodic_0到_BC_periodic_9，即6100-6109
        if (iElementR != -1) {
            // 周期和内部 统一处理
            get_UR_inner_and_periodic_device(element_device, elementField_device, edge_device, iEdge, iElementR, x_edge, y_edge, U_R, inviscid_flux_method_flag_reconstruct, gamma);
        }
    }

    get_flux_RiemannSolve_device(U_L, U_R, nx, ny, length, flux, flux_conservation_scheme, gamma, rcpcv);

    for (int j = 0; j < 4; j++) {
        edgeField_device.Flux[j][iEdge] = flux[j];
    }
}

__global__ void add_edge_flux_to_element_flux_kernel(GPU::ElementSoA element_device, GPU::ElementFieldSoA elementField_device, GPU::EdgeSoA edge_device, GPU::EdgeFieldSoA edgeField_device) {
    
    const int iElement = blockIdx.x * blockDim.x + threadIdx.x;
    if (iElement >= element_device.num_element || iElement < 0) return;

    myfloat volumeC = element_device.volume[iElement];// 体积
    // 初始化edgeID 超出数组范围的非法值赋为-1
    int edgeID[4]{ -1,-1,-1,-1 };// 第iElement个element的第i条边的ID
    for (int i = 0; i < 4; i++) {
        edgeID[i] = element_device.edges[i][iElement];
        if (edgeID[i] < 0 || edgeID[i] >= edge_device.num_edge) { // has: iEdge >= 0 && iEdge < num_edge
            edgeID[i] = -1;
        }
    }
    // 初始化边朝向edgeSign
    int edgeSign[4]{ 0,0,0,0 };// 1表示边朝外，-1表示边朝内
    for (int i = 0; i < 4; i++) {
        int edgeIDi = edgeID[i];
        if (edgeIDi == -1) {
            continue;
        }
        if (iElement != edge_device.elementR[edgeIDi]) {
            edgeSign[i] = 1;// currentElement=elementL，朝外
        }
        else {
            edgeSign[i] = -1;// currentElement=elementR，朝内
        }
    }
    // 根据边朝向，加减到中心单元。向外为加，向内为减
    for (int i = 0; i < 4; i++) {
        int edgeIDi = edgeID[i];
        if (edgeIDi == -1) {
            continue;
        }
        for (int jVar = 0; jVar < 4; jVar++) {
            elementField_device.Flux[jVar][iElement] += edgeSign[i] * edgeField_device.Flux[jVar][edgeIDi];
        }
    }
}

// 计算无粘通量
void edge_convection_flux_device() {
    GPU::GPUSolver2* solver = SolverDataGetter::getSolverInstance();
    GPU::ElementSoA& element_device = solver->element_device;
    GPU::ElementFieldSoA& elementField_device = solver->elementField_device;
    GPU::EdgeSoA& edge_device = solver->edge_device;
    GPU::EdgeFieldSoA& edgeField_device = solver->edgeField_device;
    GPU::BoundarySetMap& boundary_device = solver->boundary_device;

    int flux_conservation_scheme = GlobalPara::inviscid_flux_method::flux_conservation_scheme;
    int inviscid_flux_method_flag_reconstruct = GlobalPara::inviscid_flux_method::flag_reconstruct;
    myfloat* inf_ruvp = nullptr;
    myfloat* inlet_ruvp = nullptr;
    myfloat* outlet_ruvp = nullptr;
    hipMalloc(&inf_ruvp, 4 * sizeof(myfloat));
    hipMalloc(&inlet_ruvp, 4 * sizeof(myfloat));
    hipMalloc(&outlet_ruvp, 4 * sizeof(myfloat));
    hipMemcpy(inf_ruvp, GlobalPara::boundaryCondition::_2D::inf::ruvp, 4 * sizeof(myfloat), hipMemcpyHostToDevice);
    hipMemcpy(inlet_ruvp, GlobalPara::boundaryCondition::_2D::inlet::ruvp, 4 * sizeof(myfloat), hipMemcpyHostToDevice);
    hipMemcpy(outlet_ruvp, GlobalPara::boundaryCondition::_2D::outlet::ruvp, 4 * sizeof(myfloat), hipMemcpyHostToDevice);
    myfloat gamma = GlobalPara::constant::gamma;
    myfloat rcpcv = GlobalPara::constant::R;
    // 以下为双马赫反射的成员函数
    CBoundaryDoubleShockReflect* cbdsr = CBoundaryDoubleShockReflect::getInstance();
    myfloat dsr_shock_x = cbdsr->get_shock_x();
    myfloat dsr_shock_y = cbdsr->get_shock_y();
    myfloat dsr_shock_speed = cbdsr->get_shock_speed();
    myfloat dsr_t_RK = cbdsr->get_t_plus_dt();
    const myfloat _sqrt3 = sqrt(3.0);

    int block_size = GPU::MY_BLOCK_SIZE / 2;
    int grid_size = (edge_device.num_edge + block_size - 1) / block_size;
    dim3 block(block_size, 1, 1);
    dim3 grid(grid_size, 1, 1);
    edge_convection_flux_device_kernel <<<grid, block>>> (element_device, elementField_device, edge_device, edgeField_device, boundary_device,
        flux_conservation_scheme, inviscid_flux_method_flag_reconstruct, inf_ruvp, inlet_ruvp, outlet_ruvp, gamma, rcpcv, dsr_shock_x, dsr_shock_y, dsr_shock_speed,
        dsr_t_RK, _sqrt3);
    hipFree(inf_ruvp);
    hipFree(inlet_ruvp);
    hipFree(outlet_ruvp);
    getLastCudaError("edge_convection_flux_device failed.");
}

void add_edge_flux_to_element_flux_device() {
    GPU::GPUSolver2* solver = SolverDataGetter::getSolverInstance();
    GPU::ElementSoA& element_device = solver->element_device;
    GPU::ElementFieldSoA& elementField_device = solver->elementField_device;
    GPU::EdgeSoA& edge_device = solver->edge_device;
    GPU::EdgeFieldSoA& edgeField_device = solver->edgeField_device;
    int block_size = GPU::MY_BLOCK_SIZE;
    int grid_size = (element_device.num_element + block_size - 1) / block_size;
    dim3 block(block_size, 1, 1);
    dim3 grid(grid_size, 1, 1);
    add_edge_flux_to_element_flux_kernel <<<grid, block >>> (element_device, elementField_device, edge_device, edgeField_device);
    getLastCudaError("add_edge_flux_to_element_flux_device failed.");
}

void GPU::Space::Flux::calculateFluxDevice_2(ElementSoA& element_device, EdgeSoA& edge_device, ElementFieldSoA& elementField_device) {
    /*
    https://forums.developer.nvidia.com/t/synchronization-between-kernel-calls/23336
    两次核函数调用之间无需加hipDeviceSynchronize().
    */

    clear_element_flux_device();
    edge_convection_flux_device();
    if (GlobalPara::physicsModel::equation == _EQ_NS) GPU::Space::edge_viscous_flux_device();
    add_edge_flux_to_element_flux_device();

    getLastCudaError("GPU::Space::Flux::calculateFluxDevice_2 failed.");
}
