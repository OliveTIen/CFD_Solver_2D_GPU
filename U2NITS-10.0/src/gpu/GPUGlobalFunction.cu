#include "GPUGlobalFunction.h"
#include <iostream>
#include <sstream>
#include <ios>
#include "../output/LogWriter.h"
#include "../global/CExit.h"
#include "Env.h"

void GPU::catchCudaErrorAndExit() {
	hipError_t cuda_error = hipGetLastError();
	if (cuda_error != hipSuccess) {
		std::string e = "hipError_t=" + std::to_string(cuda_error) + ", " + hipGetErrorString(cuda_error);
		LogWriter::logAndPrint(e, LogWriter::Error, LogWriter::Error);
		CExit::saveAndExit(cuda_error);
	}
}

void __getLastCudaError(const char* errorMessage, const char* file, const int line) {
	// �μ�cuda samples��Ŀ�е�hip/hip_runtime_api.h
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		//// ԭfprintf���
		//fprintf(stderr,
		//	"%s(%i) : getLastCudaError() CUDA error :"
		//	" %s : (%d) %s.\n",
		//	file, line, errorMessage, static_cast<int>(err),
		//	hipGetErrorString(err));
		// Log
		std::stringstream ss;
		ss << file << "(" << line << ")" << " : CUDA error : " << errorMessage << " : (" << static_cast<int>(err) << ") " << hipGetErrorString(err) << ".\n";
		LogWriter::logAndPrintError(ss.str());
		exit(EXIT_FAILURE);
	}
}
