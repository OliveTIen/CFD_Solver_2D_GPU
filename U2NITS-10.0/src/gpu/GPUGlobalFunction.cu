#include "GPUGlobalFunction.h"
#include <iostream>
#include <ios>
#include "../output/LogWriter.h"

void GPU::catchCudaErrorAndExit() {
	hipError_t cuda_error = hipGetLastError();
	if (cuda_error != 0) {
		std::string e = "hipError_t=" + std::to_string(cuda_error) + ", " + hipGetErrorString(cuda_error);
		LogWriter::logAndPrint(e, LogWriter::Error, LogWriter::Error);
		exit(cuda_error);
	}
}
