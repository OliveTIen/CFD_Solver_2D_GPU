#include "GPUSolver.h"
#include "../FVM_2D.h"
#include "GPU_space.h"


void GPUSolver::initialze() {
	/*
	��FVM_2D����ת��ΪElement�ṹ����ʼ��
	��ʼ���������У�
	    ��Ԫ���ꡢU����Ԫ�ھ����ꡢ��Ԫ�ھ�U
	��ʱδ��ʼ����
		��Ԫ�ڵ����ꡢ��

	��ȡ�ļ��󣬽�FVM_2D�����ݽṹת��ΪElement�ṹ��
	1. ��ʼ��selfxy, node1, node2, node3 �������������⣬self����CPU�ϼ���ģ��Ժ��ת�Ƶ�GPU�ϣ�
	2. ��ʼ��selfValue.U1~U4, neighbor1.U1~U4, neighbor2.U1~U4, neighbor3.U1~U4
	3. �ݶȳ�ʼ��Ϊ0
	*/
	/*
hipMallocӦ����ѭ���⣬�Լ��ٿ��� �μ���P364
���Ǽ��㵥Ԫ�ͼ���������ײ�ͬ��С�����飬�����Ҫ��������
�������Ƶ��豸

��Ҫ��ʼ����
	��Ԫ����
	��Ԫ�ڵ�����

*/
	// ����GPU
	int iDeviceCount = 0;
	hipError_t error = hipGetDeviceCount(&iDeviceCount);
	if (error != hipSuccess || iDeviceCount <= 0) {
		printf("No GPU found\n");
		throw error;
	}
	printf("Num of GPU: %d\n", iDeviceCount);

	int iDev = 0;
	error = hipSetDevice(iDev);
	if (error != hipSuccess) {
		printf("Fail to set GPU %d\n", iDev);
		throw error;
	}
	printf("Activate GPU: %d\n", iDev);

	// �����ڴ�
	FVM_2D* pFVM2D = FVM_2D::pFVM2D;
	int num_element = pFVM2D->elements.size();
	this->element_host.alloc(num_element);
	
	try{ 
		element_device.cuda_alloc(num_element); 
	}
	catch (const char* e) {
		// ! �쳣����δ���
		std::cout << e << std::endl;
		hipError_t error = hipError_t::hipErrorInvalidContext;
		throw error;
	}
	
	// ��ʼ��host����
	#pragma omp parallel for
	for (int i = 0; i < num_element; i++) {
		
		Element_2D& element_i = pFVM2D->elements[i];

		// ��ʼ����Ԫ���ꡢ��ԪU����Ԫ�ݶ�
		element_host.self.isNull[i] = false;
		element_host.self.x[i] = element_i.x;
		element_host.self.y[i] = element_i.y;
		element_host.self.U1[i] = element_i.U[0];
		element_host.self.U2[i] = element_i.U[1];
		element_host.self.U3[i] = element_i.U[2];
		element_host.self.U4[i] = element_i.U[3];
		element_host.self.Ux1[i] = 0;
		element_host.self.Uy1[i] = 0;
		element_host.self.Ux2[i] = 0;
		element_host.self.Uy2[i] = 0;
		element_host.self.Ux3[i] = 0;
		element_host.self.Uy3[i] = 0;
		element_host.self.Ux4[i] = 0;
		element_host.self.Uy4[i] = 0;		

		// ��ʼ���ھ����ꡢ�ھ�U���ھ��ݶ�
		const int num_neighbor = 3;
		std::vector<Element_2D*> neighbors_element_i = element_i.findNeighbor();// �ж��ھ��Ƿ�Ϊnullptr������ֵ���˴�Ĭ��Ϊ3���ھ� ! ����������
		for (int j = 0; j < num_neighbor; j++) {
			if (neighbors_element_i[j] == nullptr) {
				element_host.neighbors[j].isNull[i] = true;
			}
			else {
				element_host.neighbors[j].isNull[i] = false;
				element_host.neighbors[j].x[i] = neighbors_element_i[j]->x;
				element_host.neighbors[j].y[i] = neighbors_element_i[j]->y;
				element_host.neighbors[j].U1[i] = neighbors_element_i[j]->U[0];
				element_host.neighbors[j].U2[i] = neighbors_element_i[j]->U[1];
				element_host.neighbors[j].U3[i] = neighbors_element_i[j]->U[2];
				element_host.neighbors[j].U4[i] = neighbors_element_i[j]->U[3];

				element_host.neighbors[j].Ux1[i] = 0;
				element_host.neighbors[j].Uy1[i] = 0;
				element_host.neighbors[j].Ux2[i] = 0;
				element_host.neighbors[j].Uy2[i] = 0;
				element_host.neighbors[j].Ux3[i] = 0;
				element_host.neighbors[j].Uy3[i] = 0;
				element_host.neighbors[j].Ux4[i] = 0;
				element_host.neighbors[j].Uy4[i] = 0;
			}
		}
	}

}

void GPUSolver::iteration() {
	/*
    
	1. ��ʼ��
		��Ԫ��ֵͨ����Ϊ0
		���ݵ�ԪU��ʼ����Ԫ�ھ�U

    2. ���㵥Ԫ�ݶ� - GPU
	    2.1
        �������ꡢU�������ݶ�
        �����������ݶ�
        ����쳣ֵ
        ���룺��Ԫ���ꡢ��ԪU����Ԫ�ھ����ꡢ��Ԫ�ھ�U
        �������Ԫ�ݶ�

	    2.2. ��Ԫ�ݶȸ��Ƶ�CPU��Ȼ������ھ��ݶ�
		
    4. ����߽���ֵͨ��
        ����߷�����nx ny��������edgex, edgey���߳�edgeLength
		    ��Ҫ�������ڵ�����
        ��������ҵ�Ԫ�ڱ߽��Uֵ(���Ҽ���UL��UR)
        ���ݱ����ͣ�����UL��UR
        ������������
            ��Ҫ�Ĳ�����UL UR nx ny edgeLength
            ������߽����ֵͨ��
    5. ���㵥Ԫ��ֵͨ��
        ���߽���ֵͨ���ӵ���Ԫ��ֵͨ���� - ��Լ������OpenMP
		(ע��ÿ��ѭ����Ҫ����Ԫ��ֵͨ����ʼ��Ϊ0)
    6. ��ʽʱ���ƽ�

 
	*/
	// ����host���ݵ�device
	element_host.cuda_copy_to_device(&element_device);

	// --- ���㵥Ԫ�ݶ� --- 
	// ���룺��Ԫ���ꡢ��ԪU����Ԫ�ھ����ꡢ��Ԫ�ھ�U
	// �������Ԫ�ݶ�
	GPU::calculateGradient(this->element_device);
	// ����device���ݵ�host
	element_device.cuda_copy_to_host(&element_host);
	// ������ Ŀǰ����Ҫ��
	
	// TODO: �����ھ��ݶ� 1.��ElementDataPack�����neighborIndex����

	// --- ����߽���ֵͨ�� --- 

}

inline void GPUSolver::finalize() {
	this->element_host.free(); 
	this->element_device.cuda_free();
}
