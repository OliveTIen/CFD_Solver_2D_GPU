#include "BoundarySetMap.h"

void GPU::BoundarySetMap::cuda_memcpy(BoundarySetMap* dist, const BoundarySetMap* src, hipMemcpyKind kind) {
	int num = dist->size;
	hipMemcpy(dist->type, src->type, num * sizeof(int), kind);
}
