#include "BoundarySetMap.h"

void GPU::BoundarySetMap::cuda_memcpy(BoundarySetMap* dist, const BoundarySetMap* src, hipMemcpyKind kind) {
	integer num = dist->size;
	hipMemcpy(dist->type, src->type, num * sizeof(integer), kind);
}
