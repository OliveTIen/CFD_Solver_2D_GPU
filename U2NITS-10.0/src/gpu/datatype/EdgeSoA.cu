#include "EdgeSoA.h"

void GPU::EdgeSoA::cuda_memcpy(EdgeSoA* dist, const EdgeSoA* src, hipMemcpyKind kind) {
	int num_edge = dist->num_edge;
	hipMemcpy(dist->ID, src->ID, num_edge * sizeof(int), kind);
	hipMemcpy(dist->nodes[0], src->nodes[0], num_edge * sizeof(int), kind);
	hipMemcpy(dist->nodes[1], src->nodes[1], num_edge * sizeof(int), kind);
	hipMemcpy(dist->setID, src->setID, num_edge * sizeof(int), kind);
	hipMemcpy(dist->elementL, src->elementL, num_edge * sizeof(int), kind);
	hipMemcpy(dist->elementR, src->elementR, num_edge * sizeof(int), kind);
	hipMemcpy(dist->length, src->length, num_edge * sizeof(REAL), kind);
	hipMemcpy(dist->distanceOfElements, src->distanceOfElements, num_edge * sizeof(REAL), kind);
	hipMemcpy(dist->xy[0], src->xy[0], num_edge * sizeof(REAL), kind);
	hipMemcpy(dist->xy[1], src->xy[1], num_edge * sizeof(REAL), kind);
	hipMemcpy(dist->normal[0], src->normal[0], num_edge * sizeof(REAL), kind);
	hipMemcpy(dist->normal[1], src->normal[1], num_edge * sizeof(REAL), kind);

}
