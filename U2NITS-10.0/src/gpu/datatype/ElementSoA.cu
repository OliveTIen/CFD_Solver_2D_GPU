#include "ElementSoA.h"



void GPU::ElementSoA::cuda_memcpy(ElementSoA* dist, const ElementSoA* src, hipMemcpyKind kind) {
	integer num_element = dist->num_element;
	hipMemcpy(dist->ID, src->ID, num_element * sizeof(integer), kind);
	hipMemcpy(dist->xy[0], src->xy[0], num_element * sizeof(REAL), kind);
	hipMemcpy(dist->xy[1], src->xy[1], num_element * sizeof(REAL), kind);
	hipMemcpy(dist->volume, src->volume, num_element * sizeof(REAL), kind);
	for (integer i = 0; i < 4; i++) {
		hipMemcpy(dist->nodes[i], src->nodes[i], num_element * sizeof(integer), kind);
		hipMemcpy(dist->edges[i], src->edges[i], num_element * sizeof(integer), kind);
		hipMemcpy(dist->neighbors[i], src->neighbors[i], num_element * sizeof(integer), kind);
		//hipMemcpy(dist->U[i], src->U[i], num_element * sizeof(REAL), kind);
		//hipMemcpy(dist->Ux[i], src->Ux[i], num_element * sizeof(REAL), kind);
		//hipMemcpy(dist->Uy[i], src->Uy[i], num_element * sizeof(REAL), kind);
		//hipMemcpy(dist->Flux[i], src->Flux[i], num_element * sizeof(REAL), kind);
	}
	
}
/*

	��Ҫ���Ƶı�����
		int num_element;

		int* ID;
		int* nodes[4];
		int* edges[4];
		int* neighbors[4];
		REAL* x;
		REAL* y;
		REAL* U[4];
		REAL* Ux[4];
		REAL* Uy[4];
		REAL* Flux[4];

	�������գ�
	int num_edge = src->num_edge;
	hipMemcpy(dist->ID, src->ID, num_edge * sizeof(int), kind);
	hipMemcpy(dist->nodes[0], src->nodes[0], num_edge * sizeof(int), kind);
	hipMemcpy(dist->nodes[1], src->nodes[1], num_edge * sizeof(int), kind);
	hipMemcpy(dist->setID, src->setID, num_edge * sizeof(int), kind);
	hipMemcpy(dist->elementL, src->elementL, num_edge * sizeof(int), kind);
	hipMemcpy(dist->elementR, src->elementR, num_edge * sizeof(int), kind);
	hipMemcpy(dist->length, src->length, num_edge * sizeof(REAL), kind);
	hipMemcpy(dist->distanceOfElements, src->distanceOfElements, num_edge * sizeof(REAL), kind);

*/

