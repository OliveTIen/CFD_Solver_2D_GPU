#include "FieldSoA.h"
#include "../Env.h"

void GPU::ElementFieldSoA::cuda_memcpy(ElementFieldSoA* dist, const ElementFieldSoA* src, hipMemcpyKind kind) {
	myint _num = dist->num;
	for (int i = 0; i < 4; i++) {
		hipMemcpy(dist->U[i], src->U[i], _num * sizeof(myfloat), kind);
		hipMemcpy(dist->Ux[i], src->Ux[i], _num * sizeof(myfloat), kind);
		hipMemcpy(dist->Uy[i], src->Uy[i], _num * sizeof(myfloat), kind);
		hipMemcpy(dist->Flux[i], src->Flux[i], _num * sizeof(myfloat), kind);
	}
}


void GPU::EdgeFieldSoA::cuda_memcpy(EdgeFieldSoA* dist, const EdgeFieldSoA* src, hipMemcpyKind kind) {
	myint _num = dist->num_edge;
	for (int i = 0; i < 4; i++) {
		hipMemcpy(dist->Flux[i], src->Flux[i], _num * sizeof(myfloat), kind);
	}
}
