#include "hip/hip_runtime.h"
#include "ReduceGPU.h"
#include "../gpu/GPUGlobalFunction.h"
#include "../math/BasicAlgorithmGPU.h"

/*
��2��kernel���ϲ�ȫ���ڴ�
ÿ���̴߳����������ݣ����������ݵĿռ���루������stride������ͬ�ģ����ƻ�������
*/
__global__ void reduce_device_kernel_1(float* input, float* output, unsigned int n, func_bin_myfloat p_func) {
    // Determine this thread's various ids
    unsigned int block_size = blockDim.x;
    unsigned int thread_id = threadIdx.x;
    unsigned int block_id = blockIdx.x;

    /*
    ÿ���̲߳���input[block_start]��input[block_start + stride]������input[block_start]
    ��˷�����ֱ�����ʣ��1����
    */
    unsigned int block_start = block_id * block_size * 2 + thread_id;// ��ǰ�̵߳��������
    for (unsigned int stride = block_size; stride > 0; stride /= 2) {
        if (thread_id < stride && // ��ǰ�̵߳���������ڷ�Χ�� 
            block_start + stride < n) // ��ǰ�̵߳��Ҳ������ڷ�Χ��
        {
            //input[block_start] += input[block_start + stride];

            p_func(input[block_start], input[block_start + stride]);
        }
        // ͬ�������߳�
        __syncthreads();
    }

    // �����ʣ�µ�1��������output
    if (!thread_id) {
        output[block_id] = input[block_start];
    }
}

//__device__ func_bin_myfloat p_operator_min = GPU::Math::operator_min;

void GPU::Math::reduce_device(const myint n, myfloat* dev_input, myfloat* dev_output, bool debug_info, ReduceType reduceType) {
    /*
   n dev_input���ܳ��ȣ�Ҫ��Ϊ2���ݡ�������ĳһ������nΪ����ʱ�����һ��Ԫ��û�в�������
   dev_input ����Լ���� ��СΪn
   dev_output �м����鼰����������� ��СΪ n/block_size ����ȡ���������Ѿ������
   */

    // ���n�Ƿ�Ϊ2���ݡ����� https://blog.csdn.net/qq_39360985/article/details/78628550
    if ((n & n - 1) == 0) {
        //printf("%d is pow of 2\n", n);
    }
    else {
        printf("warning: %d is NOT pow of 2\n", n);
    }

    func_bin_myfloat p_func_host;// reduce kernel���õ��ĺ�����ָ��

    switch (reduceType) {
    case reduceType_min:
        hipMemcpyFromSymbol(&p_func_host, HIP_SYMBOL(GPU::Math::p_operator_min), sizeof(func_bin_myfloat));// devic to host
        break;
    default:
        hipMemcpyFromSymbol(&p_func_host, HIP_SYMBOL(GPU::Math::p_operator_min), sizeof(func_bin_myfloat));
    }

    getLastCudaError("hipMemcpyFromSymbol, reduce_device failed.");

    const int block_threads = GPU::get_max_threads_per_block();
    unsigned int threads_needed = n / 2; // we'll need one thread to add every 2 elements
    unsigned int blocks = threads_needed / block_threads +  // we'll need this many blocks
        (threads_needed % block_threads > 0 ? 1 : 0); // plus one extra if threads_needed

    unsigned int remaining = n; // tracks number of elements left to add
    while (remaining > 1) {
        if (debug_info) {
            printf("Launching kernels:\n");
            printf("remaining: %u\n", remaining);
            printf("blocks: %u\n", blocks);
            printf("threads_needed: %u\n", threads_needed);
            printf("\n");
        }

        // call the kernel
        reduce_device_kernel_1 <<<blocks, block_threads>>> (dev_input, dev_output, remaining, p_func_host);

        // re-compute our size information for the next iteration
        remaining = blocks; // After the previous kernel call, each block has reduced its chunk down to a single partial sum
        threads_needed = remaining / 2; // each thread added 2 elements
        blocks = threads_needed / block_threads + (threads_needed % block_threads ? 1 : 0); // again, might need one extra block if threads_needed
        // is not evenly divisible by block_threads

        // ����ָ�롣��û�д������ݵ�host�������޿���
        if (remaining > 1) {
            float* dev_temp = dev_input;
            dev_input = dev_output;
            dev_output = dev_temp;
        }
    }
}
