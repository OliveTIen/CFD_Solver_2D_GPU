#include "hip/hip_runtime.h"
#include "EvolveGPU.h"
#include "../global/GlobalPara.h"
#include "../gpu/GPUGlobalFunction.h"
#include "../space/FluxGPU.h"
#include "../space/Flux.h"
#include "../output/LogWriter.h"
#include "../space/gradient/GradientGPU.h"
#include <sstream>

void checkCudaErrorAndExit(const char* file, int line, hipError_t err) {
    if (err != hipSuccess) {
        std::stringstream ss;
        ss << "failed at " << file << ", line " << line << ", error code " << hipGetErrorString(err) << "\n";
        LogWriter::logAndPrintError(ss.str());
        exit(EXIT_FAILURE);
    }
}

__global__ void TimeIntegration_1_kernel(GPU::DReal& dt, GPU::ElementSoA& element, GPU::ElementFieldSoA& elementField) {
    // ��ȡid���ж��Ƿ���Ч
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int id = bid * blockDim.x + tid;
    const int& ie = id;
    if (ie >= element.num_element || ie < 0) return;

    myfloat omega = element.volume[ie];
    for (int j = 0; j < 4; j++) {
        elementField.U[j][ie] -= *(dt.ptr) / omega * elementField.Flux[j][ie];
    }

}

void TimeIntegration_1(myfloat dt, GPU::ElementSoA& element_device, GPU::ElementFieldSoA& elementField_device) {
    GPU::DReal dt_device(&dt);

    int block_size = 512;// �����128 256 512
    int grid_size = (element_device.num_element + block_size - 1) / block_size;
    dim3 block(block_size, 1, 1);
    dim3 grid(grid_size, 1, 1);
    TimeIntegration_1_kernel << <grid, block >> > (dt_device, element_device, elementField_device);

}

void EvolveExplicitDevice_1(myfloat dt, GPU::ElementSoA& element_device, GPU::ElementFieldSoA& elementField_device, GPU::EdgeSoA& edge_device, GPU::BoundarySetMap& boundary, GPU::SDevicePara& para) {
    // ��ֵͨ��
    GPU::Space::Flux::calculateFluxDevice(element_device, elementField_device, edge_device, boundary, para);
    hipDeviceSynchronize();
    GPU::catchCudaErrorAndExit();

    // ʱ�����
    TimeIntegration_1(dt, element_device, elementField_device);

}

void GPU::Time::EvolveDevice(myfloat dt, int flag_timeAdvance, ElementSoA& element_device, ElementFieldSoA& elementField_device, EdgeSoA& edge_device, BoundarySetMap& boundary, SDevicePara& para) {
    if (flag_timeAdvance == _EVO_explicit) {
        EvolveExplicitDevice_1(dt, element_device, elementField_device, edge_device, boundary, para);
    }
    else {
        LogWriter::logAndPrintError("Error: invalid evolve method.\n");
        exit(1);
    }
}

__global__ void cuda_vector_divide_by_elements_with_weight_kernel(integer length, myfloat* dist, const myfloat* src, myfloat weight) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < length) {
        dist[i] /= weight * src[i];// �������ȼ� �����Ҳ�˷�������*=��/=
    }
}

void cuda_vector_divide_by_elements_with_weight(integer length, myfloat* dist, const myfloat* src, myfloat weight) {
    // ��Ȩ��������ӦԪ�����
    int threadsPerBlock = 256;
    int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;
    cuda_vector_divide_by_elements_with_weight_kernel <<<blocksPerGrid, threadsPerBlock >>> (length, dist, src, weight);
    checkCudaErrorAndExit(__FILE__, __LINE__, hipGetLastError());
}

void calculateFunctionF_modify_flux_by_volume(integer num,  myfloat* ynp_flux[4], myfloat* element_volume) {

    for (int j = 0; j < 4; j++) {
        //for (integer ie = 0; ie < num; ie++) {
        //    myfloat minus_one_on_volume = -1.0 / element_volume[ie];
        //    ynp_flux[j][ie] = minus_one_on_volume * ynp_flux[j][ie];
        //}
        cuda_vector_divide_by_elements_with_weight(num, ynp_flux[j], element_volume, -1.0);
    }
}

void calculateFunctionF(GPU::ElementSoA& element_device, GPU::NodeSoA& node_device, GPU::EdgeSoA& edge_device, GPU::ElementFieldSoA& ynp, GPU::BoundarySetMap& boundary, GPU::SDevicePara& para) {
    // ���㳣΢�ַ��̵��Ҷ���f=f(t,U)����ʱ���޹أ���˼�Ϊf(U)

    // ����U����Ux��Uy�����ع�
    GPU::Space::Gradient::Gradient_2(element_device, ynp, edge_device);
    // ����U��Ux��Uy����ͨ��������ynp.Flux
    GPU::Space::Flux::calculateFluxDevice_2(element_device, ynp, edge_device, boundary, para);
    // ����������������õ��Ҷ���f������ynp.Flux
    calculateFunctionF_modify_flux_by_volume(element_device.num_element, ynp.Flux, element_device.volume);

/*
  dim3 grid((dx / TILEX) + (!(dx % TILEX) ? 0 : 1),
            (dy / TILEY) + (!(dy % TILEY) ? 0 : 1));
  dim3 tids(TIDSX, TIDSY);

  updateVelocity_k<<<grid, tids>>>(v, vx, vy, dx, pdx, dy, TILEY / TIDSY,
                                   tPitch);
  getLastCudaError("updateVelocity_k failed.");
*/
}

__global__ void cuda_vector_add_with_weight_kernel(integer length, myfloat* dist, const myfloat* src, myfloat weight) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < length) {
        dist[i] += weight * src[i];
    }
}

void cuda_vector_add_with_weight(integer length, myfloat* dist, const myfloat* src, myfloat weight) {
    // ��Ȩ�������ӷ�
    int threadsPerBlock = 256;
    int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;
    cuda_vector_add_with_weight_kernel <<<blocksPerGrid, threadsPerBlock >>> (length, dist, src, weight);
    checkCudaErrorAndExit(__FILE__, __LINE__, hipGetLastError());
}

void evolveSingleStep_timeIntegration_scale2Darray(integer num, myfloat* U[4], myfloat dt, myfloat* flux[4]) {
    for (int i = 0; i < 4; i++) {
        cuda_vector_add_with_weight(num, U[i], flux[i], dt);
    }
}

void GPU::Time::evolveSingleStep_device(myfloat dt, GPU::ElementSoA& element_device, GPU::NodeSoA& node_device, GPU::EdgeSoA& edge_device, GPU::ElementFieldSoA& elementField_device, BoundarySetMap& boundary, SDevicePara& para) {

    // ����dU/dt = f(t,U)�Ҷ���
    calculateFunctionF(element_device, node_device, edge_device, elementField_device, boundary, para);
    // ʱ�����
    evolveSingleStep_timeIntegration_scale2Darray(elementField_device.num, elementField_device.U, dt, elementField_device.Flux);
}
